/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor_planar_complex.h"

#include "cutlass/util/reference/device/tensor_fill.h"

#include "cutlass/util/reference/device/gemm_planar_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"

#include "cutlass/library/handle.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  cutlass::gemm::GemmCoord problem_size;
  int batch_count;
  float alpha;
  float beta;

  std::string A;
  std::string B;
  std::string C;
  std::string accum;

  bool reference_check;
  int iterations;
  
  Options():
    help(false),
    problem_size({1024, 1024, 1024}),
    A("f32"),
    B("f32"),
    C("f32"),
    accum("f32"),
    batch_count(1),
    reference_check(true),
    iterations(20),
    alpha(1),
    beta() { }

  bool valid() {
    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("batch", batch_count);

    cmd.get_cmd_line_argument("A", A);
    cmd.get_cmd_line_argument("B", B);
    cmd.get_cmd_line_argument("C", C);
    cmd.get_cmd_line_argument("accum", accum);

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "18_cross_device_predict example\n\n"
        << "  This example uses the CUTLASS Library to execute GEMM computations.\n\n"
        << "Options:\n\n"
        << "  --help                      If specified, displays this usage statement.\n\n"
        << "  --m <int>                   GEMM M dimension\n"
        << "  --n <int>                   GEMM N dimension\n"
        << "  --k <int>                   GEMM K dimension\n"
        << "  --A <string>                Matrix A data type\n"
        << "  --B <string>                Matrix B data type\n"
        << "  --C <string>                Matrix C data type\n"
        << "  --accum <string>            Accumulator data type\n"
        << "  --batch <int>               Number of GEMM operations executed in one batch\n"
        << "  --alpha <f32>               Epilogue scalar alpha\n"
        << "  --beta <f32>                Epilogue scalar beta\n\n"
        << "  --iterations <int>          Number of profiling iterations to perform.\n\n";

    out << "\n\nExamples:\n\n"
        << "$ ./examples/18_cross_device_predict/18_cross_device_predict  --batch=7 --m=1024 --n=512 --k=1024 \\\n"
        << "     --alpha=2 --beta=0.707 --A=f16 --B=f16 --C=f16 --accum=f16 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = problem_size.product() * batch_count * 4;
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Performance test environment for planar complex
class TestbedGEMM {
public:

  //
  // Data members
  //
  void const *ptr_A;
  void const *ptr_B;
  void const *ptr_C;
  void *ptr_D;

  cutlass::library::Handle handle;

  cutlass::gemm::GemmCoord problem_size;
  int batch_count;

  //
  // Methods
  //

  TestbedGEMM(
    Options const &options
  ): 
    problem_size(options.problem_size), batch_count(options.batch_count) {}

  template <typename T_A, typename T_B, typename T_C>
  void initialize(
      cutlass::DeviceAllocation<T_A> &tensor_A,
      cutlass::DeviceAllocation<T_B> &tensor_B,
      cutlass::DeviceAllocation<T_C> &tensor_C)
  {

    uint64_t seed = 1073;

    // Use small integers to simplify correctness checking
    int scope_max = 6;
    int scope_min = -6;

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_A.get(), tensor_A.size(), seed, T_A(scope_max), T_A(scope_min), 0);

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_B.get(), tensor_B.size(), seed * 2019, T_B(scope_max), T_B(scope_min), 0);

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_C.get(), tensor_C.size(), seed * 2020, T_C(scope_max), T_C(scope_min), 0);
  }

  template <typename T>
  void reset(T tensor_ptr, std::string const &role) {
    if (role == "A") {
      tensor_ptr->reset(int64_t(problem_size.m()) * problem_size.k() * batch_count);
    } else if (role == "B") {
      tensor_ptr->reset(int64_t(problem_size.k()) * problem_size.n() * batch_count);
    } else if (role == "C") {
      tensor_ptr->reset(int64_t(problem_size.m()) * problem_size.n() * batch_count);
    } else if (role == "D") {
      tensor_ptr->reset(int64_t(problem_size.m()) * problem_size.n() * batch_count);
    }
  }

  void *allocate_matrix(
      std::string const &type_string,
      cutlass::library::NumericTypeID &type,
      std::string const &role)
  {
    void * ptr_;
    void * ret;
    uint64_t seed = 1073;

    // Use small integers to simplify correctness checking
    int scope_max = 6;
    int scope_min = -6;

    if (type_string == "f16") {
      ptr_ = new (cutlass::DeviceAllocation<cutlass::half_t>);
      reset((cutlass::DeviceAllocation<cutlass::half_t> *)(ptr_), role);
      cutlass::reference::device::BlockFillRandomUniform(
          ((cutlass::DeviceAllocation<cutlass::half_t>*)(ptr_))->get(), 
          ((cutlass::DeviceAllocation<cutlass::half_t>*)(ptr_))->size(), 
          seed, cutlass::half_t(scope_max), cutlass::half_t(scope_min), 0);
      ret = ((cutlass::DeviceAllocation<cutlass::half_t>*)(ptr_))->get();
      type = cutlass::library::NumericTypeID::kF16;
    }
    else if (type_string == "f32") {
      ptr_ = new (cutlass::DeviceAllocation<float>);
      reset((cutlass::DeviceAllocation<float> *)(ptr_), role);
      // std::cout << ((cutlass::DeviceAllocation<float> *)(ptr_))->size() << std::endl;
      cutlass::reference::device::BlockFillRandomUniform(
          ((cutlass::DeviceAllocation<float> *)(ptr_))->get(),
          ((cutlass::DeviceAllocation<float> *)(ptr_))->size(),
          seed, float(scope_max), float(scope_min), 0);
      ret = ((cutlass::DeviceAllocation<float> *)(ptr_))->get();
      // std::cout << ((cutlass::DeviceAllocation<float> *)(ptr_))->size() << std::endl;
      type = cutlass::library::NumericTypeID::kF32;
    }
    else {
      std::cout << "Invalid NumericTypeID: " << type_string 
            << ", should be one of [f16|f32]" << std::endl;
      exit(1);
    }
    return ret;
  }

  Result profile(Options const &options) {

    Result result;

    cutlass::library::NumericTypeID type_A;
    cutlass::library::NumericTypeID type_B;
    cutlass::library::NumericTypeID type_C;
    cutlass::library::NumericTypeID type_accum;

    // Allocate device memory for GEMM
    void *ptr_A = allocate_matrix(options.A, type_A, "A");
    void *ptr_B = allocate_matrix(options.B, type_B, "B");
    void *ptr_C = allocate_matrix(options.C, type_C, "C");
    void *ptr_D = allocate_matrix(options.C, type_C, "D");

    allocate_matrix(options.accum, type_accum, "null");
    // initialize(*ptr_A, *ptr_B, *ptr_C);

    int64_t batch_stride_A = int64_t(problem_size.m()) * problem_size.k();
    int64_t batch_stride_B = int64_t(problem_size.k()) * problem_size.n();
    int64_t batch_stride_C = int64_t(problem_size.m()) * problem_size.n();
    int64_t batch_stride_D = int64_t(problem_size.m()) * problem_size.n();

    using LayoutA = cutlass::layout::ColumnMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::ColumnMajor;

    int lda = LayoutA::packed({problem_size.m(), problem_size.k()}).stride(0);
    int ldb = LayoutB::packed({problem_size.k(), problem_size.n()}).stride(0);
    int ldc = LayoutC::packed({problem_size.m(), problem_size.n()}).stride(0);
    int ldd = LayoutC::packed({problem_size.m(), problem_size.n()}).stride(0);

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMMs
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {

      //
      // Execute the planar complex GEMM kernel via the CUTLASS Library's
      // dispatch routines.
      //
      // Note, for planar complex GEMM kernels, all numeric type arguments 
      // specify the data type of the base real types. These are understood to
      // apply to planar complex representations of matrices in memory and to complex<T>
      // structures for scalars.
      //
      // See tools/library/include/cutlass/library/handle.h for more details.
      //

      result.status = handle.gemm_universal(
          cutlass::library::GemmUniversalMode::kGemm,

          problem_size.m(), // GEMM M dimension
          problem_size.n(), // GEMM N dimension
          problem_size.k(), // GEMM K dimension

          type_accum, // Base data type of complex-valued accumulation
          type_accum, // Base data type of complex-valued alpha/beta scalars

          &options.alpha, // Pointer to alpha scalar, of type complex<T>

          type_A,                                       // Base data type of complex-valued A matrix
          cutlass::library::LayoutTypeID::kColumnMajor, // Layout of A matrix
          cutlass::library::ComplexTransform::kNone,    // Complex transformation on A matrix operand
          ptr_A,                                        // Pointer to A matrix in Global Memory
          lda,                                          // Leading dimension of A matrix

          type_B,                                       // Base data type of complex-valued B matrix
          cutlass::library::LayoutTypeID::kColumnMajor, // Layout of B matrix
          cutlass::library::ComplexTransform::kNone,    // Complex transformation on B matrix operand
          ptr_B,                                        // Pointer to B matrix in Global Memory
          ldb,                                          // Leading dimension of B matrix

          &options.beta, // Pointer to beta scalar, of type complex<T>

          type_C, // Base data type of complex valued C and D matrices

          ptr_C, // Pointer to C matrix
          ldc,   // Leading dimension of C matrix

          ptr_D, // Pointer to D matrix
          ldd,   // Leading dimension of D matrix

          batch_count, // Batch count or number of split-K slices

          batch_stride_A, // Batch stride of A operand
          batch_stride_B, // Batch stride of B operand
          batch_stride_C, // Batch stride of C operand
          batch_stride_D  // Batch stride of D operand

      );

      if (result.status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS internal error - configuration not supported" << std::endl;
        std::cerr << cutlass::cutlassGetStatusString(result.status) << std::endl;
        return result;
      }
    }
    
    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    if (handle.get_last_operation()) {
      std::cout << "Recently executed '" << handle.get_last_operation()->description().name << "'" << std::endl;
    }

    std::cout << "Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << " GFLOPs: " << result.gflops << std::endl;

    return result;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //
  // Volta Tensor Core operations are first available in CUDA 10.1 Toolkit.
  //
  // Turing Tensor Core operations are first available in CUDA 10.2 Toolkit.
  //

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major < 7) {
    std::cerr << "Volta Tensor Core operations must be run on a machine with compute capability at least 70."
              << std::endl;

    // Returning zero so this test passes on older architectures even though its actions are no-op.
    return 0;
  }
  else if (props.major == 7 && props.minor <= 2) {
    //
    // If running on the Volta architecture, at least CUDA 10.1 Toolkit is required to run this example.
    //
    if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 1))) {
      std::cerr << "Volta Tensor Core operations must be compiled with CUDA 10.1 Toolkit or later." << std::endl;

      // Returning zero so this test passes on older Toolkits even though its actions are no-op.
      return 0;
    }
  }
  else if (props.major == 7 && props.minor >= 5) {
    //
    // If running on the Turing architecture, at least CUDA 10.2 Toolkit is required to run this example.
    //
    if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
      std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;
    
      // Returning zero so this test passes on older Toolkits even though its actions are no-op.
      return 0;
    }
  }
  else {
    // NVIDIA Ampere Architecture GPUs (SM80 and later) are fully supported on CUDA 11 Toolkit and beyond.
    //
    // fall through
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  // Execute one problem size
  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }

  TestbedGEMM testbed(options);

  Result result = testbed.profile(options);

  return result.passed ? 0 : -1;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

